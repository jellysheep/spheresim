
#include <hip/hip_runtime.h>
#ifdef ENGINE_CUDA

#include "cuCudaCalculator.h"

#include <iostream>
using namespace std;

#include "NanosecondTimer.h"

template <int dims, bool _3D_>
cuCudaCalculator<dims,_3D_>::cu_cuCudaCalculator(){
	printf("CudaCalculator initialized!\n");
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_initSphere(int i){
}


template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_save(){
	//not yet implemented
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_calcWallResistance(){
}

/////////////////////////////////////////////////////////////////////////////////////////////////
template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_doStep(){
}
/////////////////////////////////////////////////////////////////////////////////////////////////

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_updateG(){
	//nothing to do
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_updateAirResistance(){
	//nothing to do
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_updateWallResistance(){
	//nothing to do
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_updateEModul(){
	//_E = curUnit.size*curUnit.size*E*1000000.0;
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_updatePoisson(){
	//nothing to do
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_updateElasticity(){
	//nothing to do
}

template <int dims, bool _3D_>
bool cuCudaCalculator<dims,_3D_>::cu_saveFrame(){
	return true;
}

template <int dims, bool _3D_>
bool cuCudaCalculator<dims,_3D_>::cu_isFixed(int i){
	return false;
}

#define k_b 1.3806488E-23
template <int dims, bool _3D_>
scalar cuCudaCalculator<dims,_3D_>::cu_getTemperature(){
	return 0;
}

template <int dims, bool _3D_>
Sphere* cuCudaCalculator<dims,_3D_>::cu_getDirectSphere(int i){
	return NULL;
}

template <int dims, bool _3D_>
Sphere* cuCudaCalculator<dims,_3D_>::cu_getSphere(int i){
	return NULL;
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_fpsChanged(scalar timeInt){
	timeInterval = timeInt;
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_boxSizeChanged(){
	updateGridSize();
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_updateGridSize(){
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_gravityChanged(){
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_sphereCountChanged_subclass(int i){
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_maxSphereCountChanged_subclass(int i){
}


template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_loadConfig(const char* file){
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_loadConfig(){
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_paintGL(bool b){
	Calculator::paintGL(b);
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_setGridWith(scalar s){
	gridWidth = s;
}

template <int dims, bool _3D_>
void cuCudaCalculator<dims,_3D_>::cu_updateMagnitude(){
}

template class cuCudaCalculator<2,false>;
template class cuCudaCalculator<3,true>;

#endif
